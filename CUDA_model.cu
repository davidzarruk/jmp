#include "hip/hip_runtime.h"
//
//  Created by David Zarruk Valencia on June, 2016.
//  Copyright (c) 2016 David Zarruk Valencia. All rights reserved.
//

#include <algorithm>
#include <cmath>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <math.h>
#include <sstream>
#include <string>
#include <stdio.h>
#include <vector>
#include <unistd.h>
#include <stdio.h>
#include <errno.h>
#include <gsl/gsl_sf_bessel.h>
#include <gsl/gsl_roots.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_math.h>
#include <unistd.h>
#include <omp.h>
#include <nlopt.hpp>
using std::vector;
using namespace std;

//****************************************************//
//            1. Parameters                  //
//****************************************************//


class parameters{
  public:
	int maxiter;
  int uti;
  double tol;
  double convergence;
  int T;
  int Tretirement;
  int yearspp;

  // Grid for savings: a
  int na;
  double amin;
  double amax;

  // Grid for mortgages: m
  int nm;
  double mmin;
  double mmax;

  // Grid for housing: h
  int nh;
  double hmin;
  double hmax;

  // Grid for renting: r
  int nr;
  double rmin;
  double rmax;

  // Grid for labor: l
  int nl;
  double lmax;

  // Grid for deoreciation: ddelta
  int nd;
  double dmin;
  double dmax;

  // Grid for income shocks: y
  int ny;
  double ssigma_y;
  double llambda_y;
  double m_y;

  // Preferences
  double ssigma;
  double rrho;
  double ppsi;
  double bbeta;
  double kkappa;

  double tthetalab;
  double eetalab;

  double oomega;
  double sunk;
  double interm;
  double rec_probab;

  // Equilibrium objects
  double ddeltabar_today;
  double ddeltabar_tomorrow;
  double ddeltaf;
  double r;
  double Ph_today;
  double Ph_tomorrow;
  double q;
  double Pa;
  double housing_supply;
  double fcost;
  double refcost;
  double pension;
  double sstax;
  double ltax;
  double lumpsum;
  double Atech;

  int compute_equivalent;

  double multiplier;

  double *d_rental;
  double *d_housing;

	void load(const char*);
};


//****************************************************//
//            1. Import functions                  //
//****************************************************//


#include "Colormod.h" // namespace Color
#include "grid_initialization.cpp"
#include "export_arrays.cpp"
#include "CUDA_functions.cu"
#include "main_functions.cpp"
#include "Aggregation_functions.cpp"
#include "bank_aggregates.cpp"
#include "transitions.cpp"
#include "steady_state.cpp"
#include "maximizaciones.cpp"
#include "transition_max.cpp"
#include "import_arrays.cpp"



//======================================
//         Value Function Iteration
//======================================


double mindosnum(const double a, const double b){
  double res = a;
  if(b<a){
    res = b;
  }
  return(res);
}


double maxdosnum(const double a, const double b){
  double res = a;
  if(b>a){
    res = b;
  }
  return(res);
}


int main(int argc, char *argv[])
{

	cout.precision(6);
	cout.setf(std::ios::fixed);


  //----------------------------------------------//
  //---------    SETTING PARAMETERS    -----------//
  //----------------------------------------------//

  // -------- STEADY STATE PARAMETERS   ----------//

  // VFI parameters
  double tol         = pow(10,-4.0);
  const int uti            = 1;
  int maxiter        = 10;
  const double convergence = 0.7; // Entre mas alto, mas lenta es la convergencia

  // Years per period
  double yearspp = 5;

  // Demographics
  int T;
  int Tretirement;

  T           = 30;
  Tretirement = 23;

  // Grid for savings: a
  int na       = 35;  // 25
  const double amin  = 0;
  double amax  = 1.5;

  // Grid for mortgages: m
  int nm       = 5;  // 11
  const double mmin  = 0.0;
  double mmax  = 0.5;

  // Grid for housing: h
  int nh       = 3;
  const double hmin  = 0.0;
  double hmax  = 3.5;

  // Grid for renting: r
  const int nr       = 9;
  const double rmin  = 0.0001;
  double rmax  = 3.5;

  // Grid for labor: l
  int nl        = 3;
  double lmax   = 0.4;

  // Grid for depreciation: ddelta
  int nd             = 3;
  double ssigma_eps  = 0.05;

  // Grid for income shocks: y
  int ny                  = 3;
  const double ssigma_y   = 0.1*pow(yearspp, 0.5); // 0.44
  const double llambda_y  = pow(0.98, yearspp); // pow(0.98, yearspp) ; pow(0.95, yearspp)
  double m_y              = 1; // 2

  // Preferences
  const double ssigma  = 2;
  double rrho          = 0.8;
  double ppsi          = 0.65;
  double bbeta         = pow(0.965445931, yearspp);
  const double kkappa  = -0.1;

  double Atech         = 2.0;

  double tthetalab     = 30.0;    // So steady state labor is around 0.36
  double eetalab       = 2.0;     // Rogerson - macro Frisch elasticity

  double oomega        = 0.2;
  double sunk          = 0.0;
  double interm        = 0.0;

  // Equilibrium objects
  double r              = pow(1.03, yearspp)-1; // Average of 10yr Treasury bond for 2003-2007: 2.07%
  double Pa             = 1/(1+r);

  // Optimizer initial values
  double Ph             = 1.0;
  double q              = 0.236237;
  double dmin           = -0.050488;
  double dmax           = 0.150493;
  double ddeltabar      = 0.038;
  double ddeltaf        = 1.0;
  double fcost          = 0.153209;
  double refcost        = 0.05;
  double housing_supply = 21.5;
  double pension        = 0.6;
  double sstax          = 0.1;
  double ltax           = 0.0;
  double rec_probab     = 0.0;

  double lumpsum        = 0.0;

  // Transitional dynamics' parameters
  int     Ttrans          = 0;
  int     periods_shock   = 0;
  int     periods_taxes   = periods_shock;

  double  rshock          = r;
  double  Pashock         = Pa;
  double  ppsishock       = ppsi;
  double  oomegashock     = oomega;
  double  sunk_shock      = sunk;
  double  interm_shock    = interm;
  double  ltaxshock       = ltax;
  double  Phinf;
  double  ddeltabarinf;
  double  qinf;
  double  interminf;

  double prob_mistake = 0.0;


  int permanent       = 0;
  int baseline        = 1;

  double multiplier       = 0.008;
  int compute_equivalent  = 0;

  double *Phseq, *Phnoextseq, *qseq, *ddeltabarseq, *lumpsumseq, *mort_subsidy, *qnoextseq, *incshock;
  int *subs_eligible, *subs_target;
  size_t sizemats, sizematssubs;
  int ind;

  // Inputs to see what to compute: initial ss, transition dynamics, etc.
  std::string stage = argv[1];
  std::string tipo  = "tr";


  int maxim = 3;

  const int experimento = 35;


  tol         = pow(10,-4.0);
  // maxiter = 20;

  // Every period: 2 years
  yearspp         = 2;
  T               = 30;
  Tretirement     = 22.5;

  Atech           = 0.8;

  // Grids
  ny              = 3;
  nd              = 3;
  na              = 11;
  nl              = 3;
  lmax            = 0.4;
  // nh = 7;
  // dmax = 0.5;

  mmax            = 0.4;
  hmax            = 3;
  rmax            = hmax;
  amax            = 1.0;

  Ph              = 1.0;
  r               = pow(1.0207, yearspp)-1; // Average of 10yr Treasury bond for 2003-2007: 2.07%
  Pa              = 1/(1+r);

  ppsi            = 0.84;
  bbeta           = pow(0.966499, yearspp);
  rrho            = 1 - yearspp/25;

  ssigma_eps      = 0.04;

  q               = 0.043131;
  dmin            = -0.349340;
  dmax            = 0.459476;
  fcost           = 0.015;
  refcost         = 0.025;

  // fcost           = 0.0;
  // refcost         = 0.0;

  ddeltabar       = 0.00;

  oomega          = -0.25;    // Esto es como Campbell y Cocco

  sunk            = 0.22;  // Mitman (pg18) de Pennington and Cross: 22%, Chatterjee tiene 17%

  interm          = pow(1.000, yearspp)-1;
  rec_probab      = 0.0;

  tthetalab       = 5.0;
  eetalab         = 0.5;

  sstax           = 0.1;
  ltax            = 0.0;

  // ltax            = 0.0035;

  pension         = 0.104;

  lumpsum         = -0.036;

  maxim = 9;

  q               = 0.047873;
  dmin            = -0.345419;
  dmax            = 0.464803;
  bbeta           = pow(0.96, yearspp);
  // r               = pow(1.023, yearspp)-1;



  // -------- TRANSITIONAL DYNAMICS PARAMETERS   ----------//

  housing_supply  = 40.65;
  ddeltabar       = 0.0;


  // TRANSITIONAL DYNAMICS - SHOCKS

  // Final steady state - same as initial
  interminf       = interm;
  ddeltabarinf    = ddeltabar;
  Phinf           = Ph;
  qinf            = q;


  // Shocking interest rate
  Ttrans          = 10;
  // rshock          = pow(1.05, yearspp)-1;
  rshock          = r;
  Pashock         = Pa;
  ppsishock       = ppsi;

  oomegashock     = 0.25;
  // oomegashock     = oomega;

  sunk_shock      = sunk;
  interm_shock    = pow(1.000, yearspp)-1;
  Pashock         = Pa;

  ltaxshock       = 0.00;
  // ltaxshock       = 0.000;

  periods_shock   = 4;
  periods_taxes   = 1;
  permanent       = 0;
  
  // Transitional dynamics objects
  sizemats        = Ttrans*sizeof(double);
  Phseq           = (double*)malloc(sizemats);
  Phnoextseq      = (double*)malloc(sizemats);
  qseq            = (double*)malloc(sizemats);
  qnoextseq       = (double*)malloc(sizemats);
  ddeltabarseq    = (double*)malloc(sizemats);
  lumpsumseq      = (double*)malloc(sizemats);

  // Subsidies only in the first period after shock
  sizematssubs    = ny*na*nm*nh*T*sizeof(double);
  mort_subsidy    = (double*)malloc(sizematssubs);

  sizematssubs    = ny*na*nm*nh*nd*T*sizeof(int);
  subs_eligible   = (int*)malloc(sizematssubs);
  subs_target     = (int*)malloc(sizematssubs);

  sizemats        = T*sizeof(double);
  incshock        = (double*)malloc(sizemats);

  // Ph
  for(int it=0; it<Ttrans; it++){
    Phseq[it]         = Ph;
    ddeltabarseq[it]  = ddeltabar;
    qseq[it]          = qinf;
    qnoextseq[it]          = qinf;
    lumpsumseq[it]    = lumpsum;
  }

  lumpsumseq[0] = -0.043;

  for(int it=0; it<T; it++){
    if(it >= 0 & it < 10/yearspp){
      incshock[it] = 0.128;
    } else if(it >= 10/yearspp && it < 20/yearspp){
      incshock[it] = 0.111;
    } else if(it >= 20/yearspp && it < 30/yearspp){
      incshock[it] = 0.088;
    } else if(it >= 30/yearspp && it < 40/yearspp){
      incshock[it] = 0.096;
    } else if(it >= 40/yearspp && it < 45/yearspp){
      incshock[it] = 0.044;
    } else{
      incshock[it] = 0.0;
    }

    incshock[it] = incshock[it]*1.0;
    // incshock[it] = incshock[it]*0.0;
  }

  Phseq[0] = 0.765;
  Phseq[1] = 0.815886;
  Phseq[2] = 0.876203;
  Phseq[3] = 0.993659;
  Phseq[4] = 0.992694;
  Phseq[5] = 0.992827;
  Phseq[6] = 0.995083;
  Phseq[7] = 0.996250;
  Phseq[8] = 0.999546;
  Phseq[9] = 0.999950;

  qnoextseq[0]  = q - 0.0057;
  qnoextseq[1]  = q - 0.0045;
  qnoextseq[2]  = q - 0.0028;
  qnoextseq[3]  = q - 0.00015;
  qnoextseq[4]  = q - 0.0001;
  qnoextseq[5]  = q - 0.00005;
  qnoextseq[6]  = q - 0.000;
  qnoextseq[7]  = q - 0.000;
  qnoextseq[8]  = q - 0.000;
  qnoextseq[9]  = q - 0.000;


  // Mortgage subsidy
  for(int it=0; it<T; it++){
    for(int iy=0; iy<ny; iy++){
      for(int ih=0; ih<nh; ih++){
        for(int im=0; im<nm; im++){
          for(int ia=0; ia<na; ia++){
            ind = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

            mort_subsidy[ind]   = 0.0;

            for(int id=0; id<nd; id++){
              ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;

              subs_eligible[ind]  = 0;
              subs_target[ind]    = 0;
            }
          }
        }
      }
    }
  }


  double *Value, *Value_future, *Policyc, *Pricing_guess, *Pcond, *Puncond;
  int *Default, *Default0, *Renew, *Policya, *Policym, *Policyh, *Policyr, *Policyl, *Changer;

  sizemats     = ny*na*nm*nh*nd*T*sizeof(double);
  size_t sizematsint  = ny*na*nm*nh*nd*T*sizeof(int);

  size_t sizematsintinfo  = ny*na*nm*nh*T*sizeof(int);
  
  Value         = (double*)malloc(sizemats);
  Value_future  = (double*)malloc(sizemats);
  Policyc       = (double*)malloc(sizemats);
  Pricing_guess = (double*)malloc(sizemats);
  Puncond       = (double*)malloc(sizemats);
  Pcond         = (double*)malloc(sizemats);

  Default0      = (int*)malloc(sizematsint);
  Default       = (int*)malloc(sizematsint);
  Renew         = (int*)malloc(sizematsint);
  Policya       = (int*)malloc(sizematsint);
  Policym       = (int*)malloc(sizematsint);
  Policyh       = (int*)malloc(sizematsint);
  Policyr       = (int*)malloc(sizematsint);
  Policyl       = (int*)malloc(sizematsint);

  Changer       = (int*)malloc(sizematsintinfo);

  int indd;

  for(int it=0; it<T; it++){
    for(int iy=0; iy<ny; iy++){
      for(int ih=0; ih<nh; ih++){
        for(int im=0; im<nm; im++){
          for(int ia=0; ia<na; ia++){
            indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

            Changer[indd]      = 0;

            for(int id=0; id<nd; id++){
              ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
              
              Value[ind]         = 0.0;
              Value_future[ind]  = 0.0;
              Policyc[ind]       = 0.0;
              Pricing_guess[ind] = 0.0;
              Puncond[ind]       = 0.0;
              Pcond[ind]         = 0.0;

              Default[ind]       = 0;
              Default0[ind]      = 0;
              Renew[ind]         = 0;
              Policya[ind]       = 0;
              Policym[ind]       = 0;
              Policyh[ind]       = 0;
              Policyr[ind]       = 0;
              Policyl[ind]       = 0;
            }
          }
        }
      }
    }
  }

  // Initialize grids
  double agrid[na];
  double dgrid[nd];
  double hgrid[nh];
  double rgrid[nr];
  double lgrid[nl];
  double mgrid[nm];
  double ygrid[ny];
  double P[ny*ny];
  double survival[T];
  double repay_coeff[T];
  double eprocess[T];

  int find_eq = 1;



	// //----------------------------------------------//
	// //---------   INITIAL STEADY STATE   -----------//
	// //----------------------------------------------//

  if (stage == "ssinitial"){

    clock_t t_start;
    clock_t *d_t_start;
    t_start = clock();
    d_t_start = &t_start;
  
    int iteraciones = 1;
    int *d_iteraciones;
    d_iteraciones =  &iteraciones;

    double min_upto = 1000.0;
    double *d_min_upto;
    d_min_upto =  &min_upto;

    double rental = 0.0;
    double *d_rental;
    d_rental = &rental;
  
    double housing = 0.0;
    double *d_housing;
    d_housing = &housing;

    // Minimum up to this point
    double qmin = q;
    double *d_q_upto;
    d_q_upto =  &qmin;

    double dmaxmin = dmax;
    double *d_dmax_upto;
    d_dmax_upto =  &dmaxmin;

    double dminmin = dmin;
    double *d_dmin_upto;
    d_dmin_upto =  &dminmin;

    double ddeltabarmin = ddeltabar;
    double *d_ddeltabar_upto;
    d_ddeltabar_upto =  &ddeltabarmin;

    double m_ymin = m_y;
    double *d_m_y_upto;
    d_m_y_upto =  &m_ymin;

    double bbetamin = bbeta;
    double *d_bbeta_upto;
    d_bbeta_upto =  &bbetamin;

    double fcostmin = fcost;
    double *d_fcost_upto;
    d_fcost_upto =  &fcostmin;
  
    double refcostmin = refcost;
    double *d_refcost_upto;
    d_refcost_upto =  &refcostmin;
  

    //Loading the structure
    pricesolver_eq_24 paramstructura  = {maxiter, uti, tol, convergence, T, Tretirement, yearspp, na, amin, amax, 
                                        nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nl, lmax, nd,
                                        ny, ssigma_y, llambda_y, m_y, ssigma, rrho, ppsi, bbeta,
                                        kkappa, tthetalab, eetalab, oomega, sunk, interm, rec_probab, ddeltabar, 
                                        ddeltaf, r, Ph, Pa, housing_supply, pension, sstax, ltax, fcost, refcost, Atech, compute_equivalent, multiplier, lumpsum,
                                        d_iteraciones, d_min_upto, d_t_start, 
                                        d_q_upto, d_dmin_upto, d_dmax_upto, d_ddeltabar_upto, d_bbeta_upto, d_fcost_upto, d_refcost_upto,
                                        d_rental, d_housing};

    //Set up the optimization algorrithm
    nlopt_opt opt;

    opt = nlopt_create(NLOPT_LN_NELDERMEAD, 3);// Dimension 2. Algoritthm cobyla    
    // opt = nlopt_create(NLOPT_GN_CRS2_LM, 4);// Dimension 2. Algoritthm cobyla    

    nlopt_set_min_objective(opt, price_zero_eq_24, &paramstructura);
    nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance

    //const double tolerance=1.0e-5;   q, fcost, dvariance
    double LB[3]={0.001, -0.35, 0.2};
    double UB[3]={0.9, -0.1, 0.6};

    nlopt_set_lower_bounds(opt, LB);
    nlopt_set_upper_bounds(opt, UB);

    nlopt_set_maxeval(opt, 5000);

    //  double init[6sion]={0.001,0.01,0.01,0.01,0.000001,0.1};
    double init[3]={0.002, 0.1, 0.1};
    nlopt_set_initial_step(opt, init);

    // Si es suficientemente pequenho el error, lo pongo en cero y paro
    nlopt_set_stopval(opt, 0.01);

    // Initialize at:
    double xtest[3] = {};
    xtest[0] = q;      // r
    xtest[1] = dmin;      // r
    xtest[2] = dmax;      // r

    //Starting the optimization algorithm
    double minf;
    nlopt_optimize(opt, xtest, &minf);

    // Resultados
    vector<double> Res;
    Res.resize(4+1);
    for(int i=0; i<4; i++){
      Res[i]=xtest[i];
    }
    Res[4]=minf;


  } else{

    double rental = 0.0;
    double *d_rental;
    d_rental = &rental;
  
    double housing = 0.0;
    double *d_housing;
    d_housing = &housing;

    parameters params = {maxiter, uti, tol, convergence, T, Tretirement, yearspp, na, amin, amax, 
                        nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nl, lmax, nd, dmin, dmin+dmax,
                        ny, ssigma_y, llambda_y, m_y, ssigma, rrho, ppsi, bbeta, kkappa, tthetalab, eetalab, 
                        oomega, sunk, interm, rec_probab, ddeltabar, ddeltabar, ddeltaf, r, Ph, Ph, q, Pa, housing_supply, 
                        fcost, refcost, pension, sstax, ltax, lumpsum, Atech, compute_equivalent, multiplier, d_rental, d_housing};

    // Fill grids
    grid_initialize(params, agrid, dgrid, hgrid, rgrid, lgrid, mgrid, ygrid, P, survival, repay_coeff, eprocess);

    // I read Policy function of first period after transition
    import_basic(Default0, Default, Renew, Policym, Policya, Policyh, Policyr, Policyl, Puncond, Pcond);


    double pipol      = 0.0;

    double costo      = 0.0;
    double targetpop  = 0.0;
    double elig       = 0.0;

    // In first period, households receive a lump-sum transfer
    for(int it=0; it<Ttrans; it++){
      lumpsumseq[it]    = lumpsum;
    }

    lumpsumseq[0]   = -0.043;

    double targetDTI;

    prob_mistake = 0.016; // Such that in equilibrium, strategic default is 10%

    if(stage == "noext"){

      // Este se corre ANTES DE BASELINE para guardar 

      clock_t t_start;
      clock_t *d_t_start;
      t_start = clock();
      d_t_start = &t_start;
    
      int iteraciones = 1;
      int *d_iteraciones;
      d_iteraciones =  &iteraciones;
    
      double min_upto = 1000.0;
      double *d_min_upto;
      d_min_upto =  &min_upto;
    

      //Loading the structure
      transitions_qs paramstructura = {maxiter, uti, tol, convergence, T, Tretirement, yearspp, na, amin, amax, nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nl, lmax,
                                      nd, dmin, dmin+dmax, ny, ssigma_y, llambda_y, m_y, 
                                      ssigma, rrho, ppsi, bbeta, kkappa, tthetalab, eetalab, oomega, sunk, interm, rec_probab, ddeltabar, ddeltaf, 
                                      r, Ph, q, Pa, housing_supply, fcost, refcost, pension, sstax, ltax, Atech, compute_equivalent, multiplier, lumpsum, Ttrans, 
                                      rshock, Pashock, ppsishock, oomegashock, sunk_shock, interm_shock, ltaxshock,
                                      Phinf, qinf, ddeltabarinf,
                                      periods_shock, periods_taxes, experimento, permanent, baseline, tipo,
                                      qnoextseq, lumpsumseq, mort_subsidy, subs_eligible, subs_target, prob_mistake, incshock,
                                      d_iteraciones, d_min_upto, d_t_start, d_rental, d_housing};


      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 10);// Dimension 2. Algoritthm cobyla    
    
      nlopt_set_min_objective(opt, transition_eq_noext, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[10]={0.60, 0.7, 0.7, 0.9, 0.9, 0.95, 0.95, 0.95, 0.95, 0.95};
      double UB[10]={1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 300);
    
      double init[10]={0.03, 0.03, 0.01, 0.01, 0.002, 0.002, 0.001, 0.001, 0.0001, 0.0001};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[10] = {};
      // Ph
      xtest[0] = Phseq[0];
      xtest[1] = Phseq[1];
      xtest[2] = Phseq[2];
      xtest[3] = Phseq[3];
      xtest[4] = Phseq[4];
      xtest[5] = Phseq[5];
      xtest[6] = Phseq[6];
      xtest[7] = Phseq[7];
      xtest[8] = Phseq[8];
      xtest[9] = Phseq[9];

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);


    } else if(stage == "base_policy"){

      // Baseline policy: TARP (45%) and HAMP (55%) are implemented

      targetDTI  = 0.275;   // In the data is 31% - this level ensures 45% in TARP and 55% in HAMP

      ltaxshock  = 0.0;
      ltax       = 0.0035;

      Phseq[0] = 0.774241;
      Phseq[1] = 0.820033;
      Phseq[2] = 0.888033;
      Phseq[3] = 0.999999;
      Phseq[4] = 0.997753;
      Phseq[5] = 0.995629;
      Phseq[6] = 0.996954;
      Phseq[7] = 0.997434;
      Phseq[8] = 0.999592;
      Phseq[9] = 0.999937;

      qnoextseq[0]  = q - 0.0056;
      qnoextseq[1]  = q - 0.0043;
      qnoextseq[2]  = q - 0.0027;
      qnoextseq[3]  = q - 0.00012;
      qnoextseq[4]  = q - 0.0001;
      qnoextseq[5]  = q - 0.00005;
      qnoextseq[6]  = q - 0.000;
      qnoextseq[7]  = q - 0.000;
      qnoextseq[8]  = q - 0.000;
      qnoextseq[9]  = q - 0.000;

      double def_tar = 0.0;

      for(int it=0; it<T; it++){
        for(int iy=0; iy<ny; iy++){
          for(int id=0; id<nd; id++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
                  indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

                  subs_eligible[ind] = 0;
                  subs_target[ind]   = 0;
                  mort_subsidy[indd] = 0.0;

                  if(im>0 && ih > 0){
                    pipol = pipol + Puncond[ind];
                  }
                  
                  tipo = "base_policy";

                  // Subsidio a los que tienen >- targetDTI para bajarlos a ese punto
                  if(im>0 && ih>0 && Policyl[ind]>0 && mgrid[im]/(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih]) > targetDTI){

                    subs_eligible[ind]  = 1;  // Elegibles los que tienen PTI > targetPTI
                    mort_subsidy[indd] = maxdosnum(0, 1-(targetDTI*(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih])/mgrid[im]));


                    if(Default[ind] == 1){
                      subs_target[ind] = 1;   // Target son los que planean hacer default

                      def_tar = def_tar + Puncond[ind]*(double)Default[ind];
                      targetpop = targetpop + Puncond[ind];
                    }

                    costo = costo + mgrid[im]*mort_subsidy[ind]*Puncond[ind];
                    elig = elig + Puncond[ind];
                  }

                  if(it == 21 && iy == 1 && ia == 10 && im == 1 && ih == 1 && id == 2){
                    cout << "El subsidio essss = " << mort_subsidy[indd] << endl;
                  }


                }
              }
            }
          }
        }
      }

      cout << "Default = " << def_tar/targetpop << endl;
      cout << endl;
      cout << "Costo = " << costo << ", Population affected = " << targetpop/pipol << ", target = " << elig/pipol << endl;
      cout << endl;

      compute_equivalent = 1;
      multiplier = 0.01;

      baseline = 0;


    } else if(stage == "subsidy_only"){

      // Expanding HAMP and eliminating TARP

      targetDTI  = 0.23;

      ltaxshock  = 0.0;
      ltax       = 0.0025;

      Phseq[0] = 0.776;
      Phseq[1] = 0.821227;
      Phseq[2] = 0.874368;
      Phseq[3] = 0.999999;
      Phseq[4] = 0.997753;
      Phseq[5] = 0.995629;
      Phseq[6] = 0.996954;
      Phseq[7] = 0.997434;
      Phseq[8] = 0.999592;
      Phseq[9] = 0.999937;

      qnoextseq[0]  = q - 0.0056;
      qnoextseq[1]  = q - 0.0043;
      qnoextseq[2]  = q - 0.0027;
      qnoextseq[3]  = q - 0.00012;
      qnoextseq[4]  = q - 0.0001;
      qnoextseq[5]  = q - 0.00005;
      qnoextseq[6]  = q - 0.000;
      qnoextseq[7]  = q - 0.000;
      qnoextseq[8]  = q - 0.000;
      qnoextseq[9]  = q - 0.000;

      double def_tar = 0.0;

      double mean_age_subsidized = 0.0;
      double mean_age_all = 0.0;

      double pip_subsidized = 0.0;
      double pip_all = 0.0;

      for(int it=0; it<T; it++){
        for(int iy=0; iy<ny; iy++){
          for(int id=0; id<nd; id++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
                  indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

                  subs_eligible[ind] = 0;
                  subs_target[ind]   = 0;
                  mort_subsidy[indd] = 0.0;

                  if(im>0 && ih > 0){
                    pipol = pipol + Puncond[ind];
                  }

                  
                  tipo = "subsidy_only";

                  // Some summary statistics
                  if(im>0 && ih>0 && Policyl[ind]>0){
                    mean_age_all = mean_age_all + agrid[ia]*Puncond[ind];
                    pip_all = pip_all + Puncond[ind];
                    if(mgrid[im]/(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih]) > targetDTI){
                      mean_age_subsidized = mean_age_subsidized + agrid[ia]*Puncond[ind];
                      pip_subsidized = pip_subsidized + Puncond[ind];
                    }

                  }


                  // Subsidio a los que tienen >- targetDTI para bajarlos a ese punto
                  if(im>0 && ih>0 && Policyl[ind]>0 && mgrid[im]/(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih]) > targetDTI){

                    subs_eligible[ind]  = 1;
                    mort_subsidy[indd] = maxdosnum(0, 1-(targetDTI*(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih])/mgrid[im]));

                    if(Default[ind]==1){
                      subs_target[ind] = 1;

                      def_tar = def_tar + Puncond[ind]*(double)Default[ind];
                      targetpop = targetpop + Puncond[ind];
                    }

                    costo = costo + mgrid[im]*mort_subsidy[ind]*Puncond[ind];
                    elig = elig + Puncond[ind];
                  }

                  if(it == 21 && iy == 1 && ia == 10 && im == 1 && ih == 1 && id == 2){
                    cout << "El subsidio essss = " << mort_subsidy[indd] << endl;
                  }

                }
              }
            }
          }
        }
      }


      cout << "age subsidies = " << mean_age_subsidized/pip_subsidized << endl;
      cout << "age all = " << mean_age_all/pip_all << endl;

      cout << "Default = " << def_tar/targetpop << endl;
      cout << endl;
      cout << "Costo = " << costo << ", Population affected = " << targetpop/pipol << ", target = " << elig/pipol << endl;
      cout << endl;


      
      compute_equivalent = 1;
      multiplier = 0.004;

      baseline = 0;


    } else if(stage == "bailout_only"){

      ltaxshock  = 0.0;
      ltax       = 0.0047;

      Phseq[0] = 0.765;
      Phseq[1] = 0.815886;
      Phseq[2] = 0.876203;
      Phseq[3] = 0.993659;
      Phseq[4] = 0.992694;
      Phseq[5] = 0.992827;
      Phseq[6] = 0.995083;
      Phseq[7] = 0.996250;
      Phseq[8] = 0.999546;
      Phseq[9] = 0.999950;

      qnoextseq[0]  = q - 0.0059;
      qnoextseq[1]  = q - 0.0047;
      qnoextseq[2]  = q - 0.0028;
      qnoextseq[3]  = q - 0.00015;
      qnoextseq[4]  = q - 0.0001;
      qnoextseq[5]  = q - 0.00005;
      qnoextseq[6]  = q - 0.000;
      qnoextseq[7]  = q - 0.000;
      qnoextseq[8]  = q - 0.000;
      qnoextseq[9]  = q - 0.000;

      for(int it=0; it<T; it++){
        for(int iy=0; iy<ny; iy++){
          for(int id=0; id<nd; id++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
                  indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

                  tipo = "bailout_only";

                  subs_eligible[ind]  = 1;
                  subs_target[ind] = 1;

                  mort_subsidy[indd]   = 0.0;

                }
              }
            }
          }
        }
      }

      compute_equivalent = 1;
      multiplier = 0.005;

      cout << endl;
      cout << "Costo = " << costo << ", Population affected = " << targetpop/pipol << ", target = " << elig/pipol << endl;
      cout << endl;

      baseline = 0;

    } else if(stage == "first_best"){

      // Mismo subsidio de HAMP pero con un componente adicional por edad
      // Elegibilidad cambia: solo se da subsidio a hogares que cambiaron de default decision after shock

      targetDTI = 0.275;

      ltaxshock       = 0.0;
      ltax       = 0.0025;

      Phseq[0] = 0.79;
      Phseq[1] = 0.84;
      Phseq[2] = 0.895;
      Phseq[3] = 0.9983;
      Phseq[4] = 0.997175;
      Phseq[5] = 0.995046;
      Phseq[6] = 0.996910;
      Phseq[7] = 0.997412;
      Phseq[8] = 0.999589;
      Phseq[9] = 0.999941;

      qnoextseq[0]  = q - 0.0055;
      qnoextseq[1]  = q - 0.0039;
      qnoextseq[2]  = q - 0.0023;
      qnoextseq[3]  = q - 0.00012;
      qnoextseq[4]  = q - 0.0001;
      qnoextseq[5]  = q - 0.00005;
      qnoextseq[6]  = q - 0.000;
      qnoextseq[7]  = q - 0.000;
      qnoextseq[8]  = q - 0.000;
      qnoextseq[9]  = q - 0.000;


      for(int it=0; it<T; it++){
        for(int iy=0; iy<ny; iy++){
          for(int id=0; id<nd; id++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
                  indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

                  if(im>0 && ih>0 && Default[ind]==1 && Default0[ind]==0){
                    Changer[indd]      = 1;
                  }
                }
              }
            }
          }
        }
      }


      // Export eligible households
      ostringstream ss;
      ss << "matrices/Eligibles.txt";
      ofstream Eligibles (ss.str().c_str());
      if (Eligibles.is_open())
      {
        for(int it=0; it<T; it++){
          for(int iy=0; iy<ny; iy++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;
                  
                  if(im>0 && ih>0 && Changer[ind] == 1){
                    Eligibles << 1 << "\n";
                  } else{
                    Eligibles << 0 << "\n";
                  }
                }
              }
            }
          }
        }
        Eligibles.close();
      }
      else cout << "Unable to open file";

      // export_eligible(T, ny, nd, na, nh, nm, Changer);

      for(int it=0; it<T; it++){
        for(int iy=0; iy<ny; iy++){
          for(int id=0; id<nd; id++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
                  indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

                  mort_subsidy[indd] = 0.0;

                  if(im>0 && ih > 0){
                    pipol = pipol + Puncond[ind];
                  }


                  tipo = "first_best";

                  // First best - perfect information

                  if(im>0 && ih>0 && Changer[indd] == 1){
                    subs_eligible[ind]  = 1;
                    mort_subsidy[indd] = 1.0*mindosnum(1.0, 1-(targetDTI*(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih])/mgrid[im]))  + 0.013*pow(T-it,1.0);

                    if(im>0 && ih>0 && Default[ind]==1 && Default0[ind]==0){
                      subs_target[ind] = 1;
                      costo = costo + mgrid[im]*mort_subsidy[ind]*Puncond[ind];
                    }
                  }
                }
              }
            }
          }
        }
      }

      compute_equivalent = 1;
      multiplier = 0.002;

      cout << endl;
      cout << "Costo = " << costo << ", Population affected = " << targetpop/pipol << ", target = " << elig/pipol << endl;
      cout << endl;

      baseline = 0;


    } else if(stage == "second_best"){

      // Modificar tamanho del subsidio de HAMP - elegibilidad es la misma

      targetDTI  = 0.275;

      ltaxshock       = 0.0;
      ltax       = 0.0038;

      Phseq[0] = 0.77;
      Phseq[1] = 0.818;
      Phseq[2] = 0.876;
      Phseq[3] = 0.998;
      Phseq[4] = 0.997175;
      Phseq[5] = 0.995046;
      Phseq[6] = 0.996910;
      Phseq[7] = 0.997412;
      Phseq[8] = 0.999589;
      Phseq[9] = 0.999941;

      qnoextseq[0]  = q - 0.006;
      qnoextseq[1]  = q - 0.0043;
      qnoextseq[2]  = q - 0.0028;
      qnoextseq[3]  = q - 0.00015;
      qnoextseq[4]  = q - 0.0001;
      qnoextseq[5]  = q - 0.00005;
      qnoextseq[6]  = q - 0.000;
      qnoextseq[7]  = q - 0.000;
      qnoextseq[8]  = q - 0.000;
      qnoextseq[9]  = q - 0.000;

      double def_tar = 0.0;

      for(int it=0; it<T; it++){
        for(int iy=0; iy<ny; iy++){
          for(int id=0; id<nd; id++){
            for(int ih=0; ih<nh; ih++){
              for(int im=0; im<nm; im++){
                for(int ia=0; ia<na; ia++){
                  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
                  indd = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;

                  mort_subsidy[indd] = 0.0;

                  if(im>0 && ih > 0){
                    pipol = pipol + Puncond[ind];
                  }

                  
                  tipo = "second_best";

                  // Subsidio a los que tienen >- targetDTI para bajarlos a ese punto
                  if(im>0 && ih>0 && Policyl[ind]>0 && mgrid[im]/(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih]) > targetDTI){

                    subs_eligible[ind]  = 1;

                    if(it < 25){
                      mort_subsidy[indd] = 1.0*mindosnum(1.0, 1-(targetDTI*(ygrid[iy]*(1-incshock[it])*eprocess[it]*lgrid[Policyl[ind]] + agrid[ia]*r + q*hgrid[ih])/mgrid[im])) + 0.05*pow(it,1.0);
                    } else{
                      mort_subsidy[indd] = 0.0;
                    }

                    if(Default[ind]==1){
                      subs_target[ind] = 1;

                      def_tar = def_tar + Puncond[ind]*(double)Default[ind];
                      targetpop = targetpop + Puncond[ind];
                    }

                    costo = costo + mgrid[im]*mort_subsidy[ind]*Puncond[ind];
                    elig = elig + Puncond[ind];
                  }

                }
              }
            }
          }
        }
      }

      cout << "Default = " << def_tar/targetpop << endl;
      cout << endl;
      cout << "Costo = " << costo << ", Population affected = " << targetpop/pipol << ", target = " << elig/pipol << endl;
      cout << endl;


      
      compute_equivalent = 1;
      multiplier = 0.004;

      baseline = 0;

    }
  } 


  if(find_eq == 1){


    clock_t t_start;
    clock_t *d_t_start;
    t_start = clock();
    d_t_start = &t_start;
  
    int iteraciones = 1;
    int *d_iteraciones;
    d_iteraciones =  &iteraciones;
  
    double min_upto = 1000.0;
    double *d_min_upto;
    d_min_upto =  &min_upto;
  
    double rental = 0.0;
    double *d_rental;
    d_rental = &rental;
  
    double housing = 0.0;
    double *d_housing;
    d_housing = &housing;
  
    // for(int it=0; it<T; it++){
    //     cout << incshock[it] << endl;
    //   }

    //Loading the structure
    transitions_qs paramstructura = {maxiter, uti, tol, convergence, T, Tretirement, yearspp, na, amin, amax, nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nl, lmax,
                                    nd, dmin, dmin+dmax, ny, ssigma_y, llambda_y, m_y, 
                                    ssigma, rrho, ppsi, bbeta, kkappa, tthetalab, eetalab, oomega, sunk, interm, rec_probab, ddeltabar, ddeltaf, 
                                    r, Ph, q, Pa, housing_supply, fcost, refcost, pension, sstax, ltax, Atech, compute_equivalent, multiplier, lumpsum, Ttrans, 
                                    rshock, Pashock, ppsishock, oomegashock, sunk_shock, interm_shock, ltaxshock,
                                    Phinf, qinf, ddeltabarinf,
                                    periods_shock, periods_taxes, experimento, permanent, baseline, tipo,
                                    qnoextseq, lumpsumseq, mort_subsidy, subs_eligible, subs_target, prob_mistake, incshock,
                                    d_iteraciones, d_min_upto, d_t_start, d_rental, d_housing};


    //Set up the optimization algorrithm
    nlopt_opt opt;
    opt = nlopt_create(NLOPT_LN_NELDERMEAD, 10);// Dimension 2. Algoritthm cobyla    
    // opt = nlopt_create(NLOPT_GN_DIRECT_L, 5);// Dimension 2. Algoritthm cobyla
    // opt = nlopt_create(NLOPT_GN_ESCH, 5);// Dimension 2. Algoritthm cobyla
    //  opt = nlopt_create(NLOPT_LN_BOBYQA, 6);// Dimension 2. Algoritthm cobyla
  
    nlopt_set_min_objective(opt, transition_eq_noext, &paramstructura);
    nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
  
    //const double tolerance=1.0e-5;   q, fcost, dvariance
    double LB[10]={0.60, 0.7, 0.7, 0.9, 0.9, 0.95, 0.95, 0.95, 0.95, 0.95};
    double UB[10]={1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0, 1.0};
    // double LB[6]={0.18,0.18,0.75, 0.85, 1, 1};
    // double UB[6]={0.32,0.32,1, 1, 1.2, 1.15};
  
    nlopt_set_lower_bounds(opt, LB);
    nlopt_set_upper_bounds(opt, UB);
  
    nlopt_set_maxeval(opt, 300);
  
    //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
    double init[10]={0.03, 0.03, 0.01, 0.001, 0.0005, 0.0005, 0.00005, 0.00005, 0.000005, 0.000005};
    nlopt_set_initial_step(opt, init);
  
    // Si es suficientemente pequenho el error, lo pongo en cero y paro
    nlopt_set_stopval(opt, 0.01);
  
    // Initialize at:
    double xtest[10] = {};
    // Ph
    xtest[0] = Phseq[0];
    xtest[1] = Phseq[1];
    xtest[2] = Phseq[2];
    xtest[3] = Phseq[3];
    xtest[4] = Phseq[4];
    xtest[5] = Phseq[5];
    xtest[6] = Phseq[6];
    xtest[7] = Phseq[7];
    xtest[8] = Phseq[8];
    xtest[9] = Phseq[9];

    //Starting the optimization algorithm
    double minf;
    nlopt_optimize(opt, xtest, &minf);
  }


}