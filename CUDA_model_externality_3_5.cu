#include "hip/hip_runtime.h"
//
//  Created by David Zarruk Valencia on June, 2016.
//  Copyright (c) 2016 David Zarruk Valencia. All rights reserved.
//

#include <algorithm>
#include <cmath>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <math.h>
#include <sstream>
#include <string>
#include <stdio.h>
#include <vector>
#include <unistd.h>
#include <stdio.h>
#include <errno.h>
#include <gsl/gsl_sf_bessel.h>
#include <gsl/gsl_roots.h>
#include <gsl/gsl_matrix.h>
#include <gsl/gsl_blas.h>
#include <gsl/gsl_math.h>
#include <unistd.h>
#include <omp.h>
#include <nlopt.hpp>
using std::vector;
using namespace std;

//****************************************************//
//            1. Parameters                  //
//****************************************************//


class parameters{
  public:
	int maxiter;
  int uti;
  double tol;
  double convergence;
  int T;

  // Grid for savings: a
  int na;
  double amin;
  double amax;

  // Grid for mortgages: m
  int nm;
  double mmin;
  double mmax;

  // Grid for housing: h
  int nh;
  double hmin;
  double hmax;

  // Grid for renting: r
  int nr;
  double rmin;
  double rmax;

  // Grid for deoreciation: ddelta
  int nd;
  double dmin;
  double dmax;

  // Grid for income shocks: y
  int ny;
  double ssigma_y;
  double llambda_y;
  double m_y;

  // Preferences
  double ssigma;
  double rrho;
  double ppsi;
  double bbeta;
  double kkappa;

  // Equilibrium objects
  double ddeltabar_today;
  double ddeltabar_tomorrow;
  double ddeltaf;
  double r;
  double Ph_today;
  double Ph_tomorrow;
  double q;
  double Pa;
  double housing_supply;
  double fcost;

  double *d_rental;
  double *d_housing;

	void load(const char*);
};


//****************************************************//
//            1. Import functions                  //
//****************************************************//


#include "Colormod.h" // namespace Color
#include "grid_initialization.cpp"
#include "export_arrays.cpp"
#include "CUDA_functions.cu"
#include "main_functions.cpp"
#include "Aggregation_functions.cpp"
#include "bank_aggregates.cpp"
#include "transitions.cpp"
#include "steady_state.cpp"
#include "maximizaciones.cpp"
#include "transition_max.cpp"



//======================================
//         Value Function Iteration
//======================================



int main(int argc, char *argv[])
{

	cout.precision(6);
	cout.setf(std::ios::fixed);

  // VFI parameters
  const double tol         = pow(10,-5.0);
  const int uti            = 1;
  const int maxiter        = 10;
  const double convergence = 0.7; // Entre mas alto, mas lenta es la convergencia

  // Demographics
  const int T        = 12;

  // Grid for savings: a
  const int na       = 35;  // 25
  const double amin  = 0;
  const double amax  = 1.2;

  // Grid for mortgages: m
  const int nm       = 5;  // 11
  const double mmin  = 0.0;
  const double mmax  = 1.5;

  // Grid for housing: h
  const int nh       = 3;
  const double hmin  = 0.0;
  const double hmax  = 5;

  // Grid for renting: r
  const int nr       = 13;
  const double rmin  = 0.0001;
  const double rmax  = 4;

  // Grid for depreciation: ddelta
  const int nd       = 3;

  // Grid for income shocks: y
  const int ny            = 5;
  const double ssigma_y   = 0.1*pow(5,0.5); // 0.44
  const double llambda_y  = pow(0.98,5.0); // pow(0.98,5.0) ; pow(0.95,5.0)
  const double m_y        = 1.5; // 2

  // Preferences
  const double ssigma  = 2;
  const double rrho    = 0.8;
  const double ppsi    = 0.65;
  const double bbeta   = pow(0.964181319,5.0);
  const double kkappa  = -0.1;

  // Equilibrium objects
  double r              = pow(1.02, 5.0)-1;
  double Pa             = 1/(1+r);
  double housing_supply = 21.5;

  // Optimizer initial values
  const double Ph        = 1.0;
  const double q         = 0.235224;
  const double dmin      = -0.067345;
  const double dmax      = 0.156301;
  double ddeltabar = 0.038;
  double ddeltaf   = 1.0;
  const double fcost     = 0.169573;

  // Transitional dynamics' parameters
  int Ttrans;
  double rshock;
  double Pashock;
  double ppsishock;
  int periods_shock;

  const int experimento   = 2;

  if(experimento == 0){

    // Shocking interest rate
    Ttrans        = 3;
    rshock        = r;
    Pashock       = Pa;
    ppsishock     = ppsi;
    periods_shock = 0;

  } else if(experimento == 1){

    // Shocking interest rate
    Ttrans        = 3;
    rshock        = pow(1.05, 5.0)-1;
    Pashock       = Pa;
    ppsishock     = ppsi;
    periods_shock = 3;

  } else if(experimento == 2){

    // Shocking ppsi
    Ttrans        	= 6;
    rshock        	= r;
    Pashock       	= Pa;
    ppsishock     	= 0.7;
    periods_shock 	= 1;
    ddeltaf 		= 0.25;
    ddeltabar 		= 0.019;

  } else if(experimento == 3){

    // Shocking ppsi
    Ttrans        = 3;
    rshock        = pow(1.02, 5.0)-1;
    Pashock       = Pa;
    ppsishock     = 0.7;
    periods_shock = 1;

  }

  // COn ddeltaf mayor
  // const double q = 0.23;
  // const double dmin =  -0.2;
  // const double dmax =  0.703288;
  // const double ddeltabar = 1.085;
  // const double ddeltaf  = 1.0;
  // const double fcost = 0.158946;


  std::string stage = argv[1];

	// //----------------------------------------------//
	// //---------   INITIAL STEADY STATE   -----------//
	// //----------------------------------------------//

  if (stage == "initial"){

    clock_t t_start;
    clock_t *d_t_start;
    t_start = clock();
    d_t_start = &t_start;
  
    int iteraciones = 1;
    int *d_iteraciones;
    d_iteraciones =  &iteraciones;

    double min_upto = 1000.0;
    double *d_min_upto;
    d_min_upto =  &min_upto;

    double rental = 0.0;
    double *d_rental;
    d_rental = &rental;
  
    double housing = 0.0;
    double *d_housing;
    d_housing = &housing;

    // Minimum up to this point
    double qmin = q;
    double *d_q_upto;
    d_q_upto =  &qmin;

    double dmaxmin = dmax;
    double *d_dmax_upto;
    d_dmax_upto =  &dmaxmin;

    double dminmin = dmin;
    double *d_dmin_upto;
    d_dmin_upto =  &dminmin;

    double ddeltabarmin = ddeltabar;
    double *d_ddeltabar_upto;
    d_ddeltabar_upto =  &ddeltabarmin;

    double m_ymin = m_y;
    double *d_m_y_upto;
    d_m_y_upto =  &m_ymin;

    double bbetamin = bbeta;
    double *d_bbeta_upto;
    d_bbeta_upto =  &bbetamin;

    double fcostmin = fcost;
    double *d_fcost_upto;
    d_fcost_upto =  &fcostmin;

    int maxim = 3;
    
    if(maxim == 1){
      //Loading the structure
      pricesolver_eq_15 paramstructura  = {maxiter, uti, tol, convergence, T, na, amin, amax, 
                                          nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nd, 
                                          ny, ssigma_y, llambda_y, ssigma, rrho, ppsi, 
                                          kkappa, ddeltabar, ddeltaf, r, Ph, Pa, housing_supply,
                                          d_iteraciones, d_min_upto, d_t_start, 
                                          d_q_upto, d_dmin_upto, d_dmax_upto, d_ddeltabar_upto, d_bbeta_upto, d_fcost_upto, d_m_y_upto,
                                          d_rental, d_housing};

      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 6);// Dimension 2. Algoritthm cobyla    
      // opt = nlopt_create(NLOPT_GN_CRS2_LM, 6);// Dimension 2. Algoritthm cobyla    
      // opt = nlopt_create(NLOPT_GN_DIRECT_L, 6);// Dimension 2. Algoritthm cobyla
      // opt = nlopt_create(NLOPT_GN_ESCH, 6);// Dimension 2. Algoritthm cobyla
      //  opt = nlopt_create(NLOPT_LN_BOBYQA, 6);// Dimension 2. Algoritthm cobyla

      nlopt_set_min_objective(opt, price_zero_eq_15, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance

      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[6]={0.22, 0.15, 0.59, 0.04, -0.2,  0.5};
      double UB[6]={0.4, 0.35, 1.05, 0.3,  -0.05, 2.0};

      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);

      nlopt_set_maxeval(opt, 5000);

      //  double init[6sion]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[6]={0.02, 0.1, 0.1, 0.05, 0.1, 0.4};
      nlopt_set_initial_step(opt, init);

      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);

      // Initialize at:
      double xtest[6] = {};
      xtest[0] = q;      // r
      xtest[1] = dmax;      // r
      xtest[2] = bbeta;      // r
      xtest[3] = fcost;      // r
      xtest[4] = dmin;      // r
      xtest[5] = m_y;      // r

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

      // Resultados
      vector<double> Res;
      Res.resize(6+1);
      for(int i=0; i<6; i++){
        Res[i]=xtest[i];
      }
      Res[6]=minf;

      cout << "Equilibrio encontrado: " << endl;
      cout << "Rental prices q = " << Res[0]<< endl;
      cout << "Dmax = " << Res[1]<< endl;
      cout << "Ddeltabar = " << ddeltabar<< endl;
      cout << "Bbeta = " << Res[2] << endl;
      cout << "Fcost = " << Res[3] << endl;
      cout << "dmin = " << Res[4] << endl;
      cout << "Minimum = " << Res[5] << endl;

    } else if(maxim == 2){
      //Loading the structure
      pricesolver_eq_16 paramstructura  = {maxiter, uti, tol, convergence, T, na, amin, amax, nm, 
                                            mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nd, dmin, dmax, ny, ssigma_y, llambda_y, 
                                            ssigma, rrho, ppsi, bbeta, kkappa, ddeltabar, ddeltaf, r, Ph, q, Pa, 
                                            housing_supply, fcost, 
                                            d_iteraciones, d_min_upto, d_t_start, 
                                            d_q_upto, d_dmin_upto, d_dmax_upto, d_ddeltabar_upto, d_bbeta_upto, d_fcost_upto, d_m_y_upto,
                                            d_rental, d_housing};

      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 1);// Dimension 2. Algoritthm cobyla    
      // opt = nlopt_create(NLOPT_GN_CRS2_LM, 6);// Dimension 2. Algoritthm cobyla    
      // opt = nlopt_create(NLOPT_GN_DIRECT_L, 6);// Dimension 2. Algoritthm cobyla
      // opt = nlopt_create(NLOPT_GN_ESCH, 6);// Dimension 2. Algoritthm cobyla
      //  opt = nlopt_create(NLOPT_LN_BOBYQA, 6);// Dimension 2. Algoritthm cobyla

      nlopt_set_min_objective(opt, price_zero_eq_16, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance

      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[1]={0.5};
      double UB[1]={2.0};

      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);

      nlopt_set_maxeval(opt, 5000);

      //  double init[6sion]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[1]={0.4};
      nlopt_set_initial_step(opt, init);

      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);

      // Initialize at:
      double xtest[1] = {};
      xtest[0] = m_y;      // r

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

      // Resultados
      vector<double> Res;
      Res.resize(1);
      for(int i=0; i<1; i++){
        Res[i]=xtest[i];
      }
      Res[1]=minf;


    } else if(maxim == 3){
      //Loading the structure
      pricesolver_eq_14 paramstructura  = {maxiter, uti, tol, convergence, T, na, amin, amax, 
                                          nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nd, 
                                          ny, ssigma_y, llambda_y, m_y, ssigma, rrho, ppsi, 
                                          kkappa, ddeltabar, ddeltaf, r, Ph, Pa, housing_supply,
                                          d_iteraciones, d_min_upto, d_t_start, 
                                          d_q_upto, d_dmin_upto, d_dmax_upto, d_ddeltabar_upto, d_bbeta_upto, d_fcost_upto, 
                                          d_rental, d_housing};

      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 5);// Dimension 2. Algoritthm cobyla    

      nlopt_set_min_objective(opt, price_zero_eq_14, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance

      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[5]={0.13, 0.15, 0.59, 0.02, -0.2};
      double UB[5]={0.35, 0.35, 1.05, 0.3,  -0.05};

      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);

      nlopt_set_maxeval(opt, 5000);

      //  double init[6sion]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[5]={0.01, 0.05, 0.01, 0.02, 0.05};
      nlopt_set_initial_step(opt, init);

      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);

      // Initialize at:
      double xtest[5] = {};
      xtest[0] = q;      // r
      xtest[1] = dmax;      // r
      xtest[2] = bbeta;      // r
      xtest[3] = fcost;      // r
      xtest[4] = dmin;      // r

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

      // Resultados
      vector<double> Res;
      Res.resize(5+1);
      for(int i=0; i<5; i++){
        Res[i]=xtest[i];
      }
      Res[5]=minf;


    } else if(maxim == 4){
      // Encuentro 1 de equilibrio
      GenEqParameters_eq paramstructura  = {maxiter, uti, tol, convergence, T,
                                          na, amin, amax, nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, nd, dmin, dmax, ny, ssigma_y, llambda_y, m_y, ssigma,
                                          rrho, ppsi, bbeta, kkappa, ddeltabar, ddeltaf, r, Ph, Pa, housing_supply, fcost, 
                                          d_iteraciones, d_min_upto, d_t_start, 
                                            d_q_upto, d_dmin_upto, d_dmax_upto, d_ddeltabar_upto, d_bbeta_upto, d_fcost_upto, d_m_y_upto,
                                            d_rental, d_housing};

      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 1);// Dimension 2. Algoritthm cobyla    

      nlopt_set_min_objective(opt, price_zero_eq, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance

      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[1]={0.22};
      double UB[1]={0.3};

      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);

      nlopt_set_maxeval(opt, 5000);

      //  double init[6sion]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[1]={0.01};
      nlopt_set_initial_step(opt, init);

      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);

      // Initialize at:
      double xtest[1] = {};
      xtest[0] = q;      // r

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    } 


  } else if(stage == "final"){

    clock_t t_start;
    clock_t *d_t_start;
    t_start = clock();
    d_t_start = &t_start;
  
    int iteraciones = 1;
    int *d_iteraciones;
    d_iteraciones =  &iteraciones;
  
    double min_upto = 1000.0;
    double *d_min_upto;
    d_min_upto =  &min_upto;
  
    double rental = 0.0;
    double *d_rental;
    d_rental = &rental;
  
    double housing = 0.0;
    double *d_housing;
    d_housing = &housing;
  
    //Loading the structure
    transitions_qs paramstructura = {maxiter, uti, tol, convergence, T, na, amin, amax, nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, 
                                    nd, dmin, dmin+dmax, ny, ssigma_y, llambda_y, m_y, 
                                    ssigma, rrho, ppsi, bbeta, kkappa, ddeltabar, ddeltaf, 
                                    r, Ph, q, Pa, housing_supply, fcost, Ttrans, rshock, Pashock, ppsishock, periods_shock,
                                    d_iteraciones, d_min_upto, d_t_start, d_rental, d_housing};


    if(Ttrans == 3){
      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 6);// Dimension 2. Algoritthm cobyla    
  
      nlopt_set_min_objective(opt, transition_eq_qs, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
  
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[6]={0.65, 0.8, 0.85, 1, 1, 1};
      double UB[6]={1.05, 1.05, 1.05, 1.2, 1.2, 1.2};
  
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
  
      nlopt_set_maxeval(opt, 300);
  
      double init[6]={0.01, 0.01, 0.01, 0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
  
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
  
      // Initialize at:
      double xtest[6] = {};
      // Ph
      xtest[0] = 0.8552047;
      xtest[1] = 0.98105;
      xtest[2] = 0.997210;
      // xtest[0] = 1.0;
      // xtest[1] = 1.0;
      // xtest[2] = 1.0;
      // ddeltabar
      xtest[3] = 1.191;
      xtest[4] = 1.15;
      xtest[5] = 1.11;
      // xtest[3] = ddeltabar;
      // xtest[4] = ddeltabar;
      // xtest[5] = ddeltabar;

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    } else if(Ttrans == 5){
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 10);// Dimension 2. Algoritthm cobyla    
    
      nlopt_set_min_objective(opt, transition_eq_qs, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[10]={0.75, 0.8, 0.85, 0.8, 0.85, 1, 1, 1, 1, 1};
      double UB[10]={1.05, 1.05, 1.05, 1.05, 1.05, 1.2, 1.2, 1.2, 1.2, 1.1};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 1000);
    
      //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[10]={0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[10] = {};
      // Ph
      xtest[0] = 0.84;
      xtest[1] = 0.981461;
      xtest[2] = 0.995;
      xtest[3] = 0.999;
      xtest[4] = 1.002579;
      // xtest[0] = 1.0;
      // xtest[1] = 1.0;
      // xtest[2] = 1.0;
      // xtest[3] = 1.0;
      // xtest[4] = 1.0;
      // ddeltabar
      xtest[5] = 1.165;
      xtest[6] = 1.084;
      xtest[7] = 1.06;
      xtest[8] = 1.045;
      xtest[9] = 1.077;
      // xtest[5] = ddeltabar;
      // xtest[6] = ddeltabar;
      // xtest[7] = ddeltabar;
      // xtest[8] = ddeltabar;
      // xtest[9] = ddeltabar;

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    } else if(Ttrans == 6){
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 12);// Dimension 2. Algoritthm cobyla    
    
      nlopt_set_min_objective(opt, transition_eq_qs, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[12]={0.75, 0.8, 0.85, 0.8, 0.85, 0.8, 0, 0, 0, 0, 0, 0};
      double UB[12]={1.05, 1.05, 1.05, 1.05, 1.05, 1.05, 0.4, 0.2, 0.2, 0.2, 0.1, 0.1};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 1000);
    
      //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[12]={0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[12] = {};
      // Ph
      xtest[0] = 0.897913;
      xtest[1] = 0.979154;
      xtest[2] = 0.997792;
      xtest[3] = 1.010844;
      xtest[4] = 1.004875;
      xtest[5] = 1.000967;
      // xtest[0] = 1.0;
      // xtest[1] = 1.0;
      // xtest[2] = 1.0;
      // xtest[3] = 1.0;
      // xtest[4] = 1.0;
      // xtest[5] = 1.0;
      // ddeltabar
      xtest[6] = 0.083;
      xtest[7] = 0.051;
      xtest[8] = 0.016;
      xtest[9] = 0.010;
      xtest[10] = 0.018;
      xtest[11] = 0.021;
      // xtest[6] = ddeltabar;
      // xtest[7] = ddeltabar;
      // xtest[8] = ddeltabar;
      // xtest[9] = ddeltabar;
      // xtest[10] = ddeltabar;
      // xtest[11] = ddeltabar;

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    } else if(Ttrans == 8){
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 16);// Dimension 2. Algoritthm cobyla    
    
      nlopt_set_min_objective(opt, transition_eq_qs, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[16]={0.75, 0.8, 0.85, 0.8, 0.85, 0.85, 0.8, 0.85, 1, 1, 1, 1, 1, 1, 1, 1};
      double UB[16]={1.05, 1.05, 1.05, 1.05, 1.05, 1.05, 1.05, 1.05, 1.2, 1.15, 1.1, 1.1, 1.1, 1.1, 1.1, 1.1};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 300);
    
      //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[16]={0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[16] = {};
      // Ph
      xtest[0] = 0.78;
      xtest[1] = 0.98;
      xtest[2] = 0.995;
      xtest[3] = 1.0;
      xtest[4] = 1.0;
      xtest[5] = 1.0;
      xtest[6] = 1.0;
      xtest[7] = 1.0;
      // ddeltabar
      xtest[8] = 1.11;
      xtest[9] = 1.07;
      xtest[10] = 1.02;
      xtest[11] = 1.02;
      xtest[12] = 1.04;
      xtest[13] = 1.05;
      xtest[14] = 1.06;
      xtest[15] = 1.076;

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);
    }

  } else if(stage == "noext"){

    clock_t t_start;
    clock_t *d_t_start;
    t_start = clock();
    d_t_start = &t_start;
  
    int iteraciones = 1;
    int *d_iteraciones;
    d_iteraciones =  &iteraciones;
  
    double min_upto = 1000.0;
    double *d_min_upto;
    d_min_upto =  &min_upto;
  
    double rental = 0.0;
    double *d_rental;
    d_rental = &rental;
  
    double housing = 0.0;
    double *d_housing;
    d_housing = &housing;
  
    //Loading the structure
    transitions_qs paramstructura = {maxiter, uti, tol, convergence, T, na, amin, amax, nm, mmin, mmax, nh, hmin, hmax, nr, rmin, rmax, 
                                    nd, dmin, dmin+dmax, ny, ssigma_y, llambda_y, m_y, 
                                    ssigma, rrho, ppsi, bbeta, kkappa, ddeltabar, ddeltaf, 
                                    r, Ph, q, Pa, housing_supply, fcost, Ttrans, rshock, Pashock, ppsishock, periods_shock,
                                    d_iteraciones, d_min_upto, d_t_start, d_rental, d_housing};

    if(Ttrans == 3){

      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 3);// Dimension 2. Algoritthm cobyla    
      // opt = nlopt_create(NLOPT_GN_DIRECT_L, 5);// Dimension 2. Algoritthm cobyla
      // opt = nlopt_create(NLOPT_GN_ESCH, 5);// Dimension 2. Algoritthm cobyla
      //  opt = nlopt_create(NLOPT_LN_BOBYQA, 6);// Dimension 2. Algoritthm cobyla
    
      nlopt_set_min_objective(opt, transition_eq_noext, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[3]={0.75, 0.85, 0.95};
      double UB[3]={1.05, 1.05, 1.05};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 300);
    
      //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[3]={0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[3] = {};
      // Ph
      xtest[0] = 0.825;
      xtest[1] = 0.985;
      xtest[2] = 1.0;

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    }  else if(Ttrans == 6){
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 6);// Dimension 2. Algoritthm cobyla    
    
      nlopt_set_min_objective(opt, transition_eq_noext, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[6]={0.75, 0.8, 0.85, 0.8, 0.85, 0.8};
      double UB[6]={1.05, 1.05, 1.05, 1.05, 1.05, 1.05};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 1000);
    
      //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[6]={0.01, 0.01, 0.01, 0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[6] = {};
      // Ph
      xtest[0] = 0.935;
      xtest[1] = 0.985;
      xtest[2] = 0.997792;
      xtest[3] = 1.0;
      xtest[4] = 1.0;
      xtest[5] = 1.0;
      // xtest[0] = 1.0;
      // xtest[1] = 1.0;
      // xtest[2] = 1.0;
      // xtest[3] = 1.0;
      // xtest[4] = 1.0;
      // ddeltabar

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    } else if(Ttrans == 5){

      //Set up the optimization algorrithm
      nlopt_opt opt;
      opt = nlopt_create(NLOPT_LN_NELDERMEAD, 5);// Dimension 2. Algoritthm cobyla    
      // opt = nlopt_create(NLOPT_GN_DIRECT_L, 5);// Dimension 2. Algoritthm cobyla
      // opt = nlopt_create(NLOPT_GN_ESCH, 5);// Dimension 2. Algoritthm cobyla
      //  opt = nlopt_create(NLOPT_LN_BOBYQA, 6);// Dimension 2. Algoritthm cobyla
    
      nlopt_set_min_objective(opt, transition_eq_noext, &paramstructura);
      nlopt_set_xtol_rel(opt, 1.0e-5); //Tolerance
    
      //const double tolerance=1.0e-5;   q, fcost, dvariance
      double LB[5]={0.75, 0.85, 0.95, 0.95, 0.95};
      double UB[5]={1.05, 1.05, 1.05, 1.05, 1.05};
      // double LB[6]={0.18,0.18,0.75, 0.85, 1, 1};
      // double UB[6]={0.32,0.32,1, 1, 1.2, 1.15};
    
      nlopt_set_lower_bounds(opt, LB);
      nlopt_set_upper_bounds(opt, UB);
    
      nlopt_set_maxeval(opt, 300);
    
      //  double init[6]={0.001,0.01,0.01,0.01,0.000001,0.1};
      double init[5]={0.01, 0.01, 0.01, 0.01, 0.01};
      nlopt_set_initial_step(opt, init);
    
      // Si es suficientemente pequenho el error, lo pongo en cero y paro
      nlopt_set_stopval(opt, 0.01);
    
      // Initialize at:
      double xtest[5] = {};
      // Ph
      xtest[0] = 0.82;
      xtest[1] = 0.97;
      xtest[2] = 0.997;
      xtest[3] = 0.998;
      xtest[4] = 1.0;

      //Starting the optimization algorithm
      double minf;
      nlopt_optimize(opt, xtest, &minf);

    }
  }

}

