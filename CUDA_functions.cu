
#include <hip/hip_runtime.h>


//======================================
//         Utility function
//======================================

__device__ double u(const double c, const double h, const double l, 
                    const double ssigma, const double ppsi, const int uti, 
                    const double kkappa, const double tthetalab, const double eetalab){
  
  double utility = 0.0; 
  
  if(uti == 1){
    // CES
    utility = (powf(powf(ppsi*powf(c, kkappa) + (1-ppsi)*powf(h, kkappa), (1/kkappa)), 1-ssigma) / (1-ssigma)) - (tthetalab*pow(l, 1 + eetalab)/(1 + eetalab)) ;
  } else if(uti == 2){
    // Utility function 2 
    utility = powf(powf(c, ppsi)*powf(h, 1-ppsi), 1-ssigma) / (1-ssigma);
  }
  
  if(c <= 0 || h <= 0){
    utility = powf(-10, 15);
  }

  return(utility);
}


__device__ double mortg_function(const double m, const double Pm, const double oomega, const double h, const double Ph, const double repay_coeff){
  
  double mortgage = 0.0; 
  
  if(m*(1+repay_coeff) <= (1-oomega)*Ph*h){
  // if(m*Pm <= (1-oomega)*Ph*h){

    mortgage = m*Pm;

  } else{

    mortgage = -10000.0;

  }

  return(mortgage);
}

__device__ double maximumab(const double a, const double b){
  
  double ans = a; 
  
  if(b >= a){
    ans = b;
  }

  return(ans);
}


//======================================
//         Pricing function
//======================================


__global__ void Pmort(const int T, const int na, const int nm, const int nh, const int nd, const int ny,
                      const double rrho, const double r, const double Ph, const double ddeltabar, const double sunk, const double interm, const double rec_probab,
                      const double *P, 
                      const double *dgrid, 
                      const double *mgrid, 
                      const double *hgrid, 
                      const double *rgrid, 
                      const double *survival,
                      const double *repay_coeff,
                      const int it,
                      const int* Def,
                      const int* Renew,
                      const int* Policya,
                      double* pricing,
                      double* pricing_guess){

  const int id  = threadIdx.x;
  const int ih  = threadIdx.y;
  const int iy  = threadIdx.z;

  const int im  = blockIdx.x;
  const int ia  = blockIdx.y;

  // If mortgage is equal to zero, the price is not relevant.
  if(im > 0){
    int ind;
    int ind2;
    int ind3;
    
    int iap = 0;
  
    // For every state variable, I compute the pricing function
    ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
  
    if(it == T-1){
  
      pricing[ind] = 0.0;
  
    } else{
  
      // Expected value is sum over tomorrow's possible shocks times probabilities: P[iy][iyp]*(1/nd)
      for(int iyp=0; iyp<ny; iyp++){
        for(int idp=0; idp<nd; idp++){
  
          ind2 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + im*na + ia;
  
          iap = Policya[ind2];
          ind3 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + im*na + iap;
          
          // pricing[ind] = pricing[ind] + ((survival[it]*rrho/(mgrid[im]*(1+r+interm)))*P[iy*ny+iyp]*(1/(double)nd)*(Def[ind2]*(1-Renew[ind2])*Ph*(1-dgrid[idp] - ddeltabar)*hgrid[ih]*(1-sunk) + // If he defaults 
          pricing[ind] = pricing[ind] + ((survival[it]*rrho/(mgrid[im]*(1+r+interm)))*P[iy*ny+iyp]*(1/(double)nd)*(Def[ind2]*(1-Renew[ind2])*(Ph*(1-dgrid[idp] - ddeltabar)*hgrid[ih]*(1-sunk) - Ph*dgrid[idp]*hgrid[ih]) + // If he defaults 
                                                                                                          (1-Def[ind2])*((1-Renew[ind2])*(mgrid[im] + pricing_guess[ind3]*mgrid[im]) + // If he pays and continues with mortgage 
                                                                                                          (1-Def[ind2])*Renew[ind2]*(mgrid[im] + repay_coeff[it+1]*mgrid[im]))));
        }
      }
    }
  }
}


//======================================
//     Value Function Computation
//======================================

__global__ void vfi(const int T, const int Tretirement, const int na, const int nm, const int nh, const int nr, const int nl,
                    const int nd, const int ny, const int uti, const double rrho, 
                    const double bbeta, const double Ph, const double q, 
                    const double Pa, const double ddeltabar, const double ssigma, 
                    const double ppsi, const double kkappa, const double tthetalab, const double eetalab,
                    const double fcost, const double refcost, const double pension, const double sstax, const double ltax,
                    const double lumpsum, const double oomega, const double rec_probab, const double sunk,
                    const double *incshock, const double *mortsubsidy,
                    const double *agrid, const double *mgrid, const double *hgrid, const double *rgrid, const double *lgrid,
                    const double *dgrid, const double *ygrid, const double *P, 
                    const double *eprocess, const double *survival, const double *repay_coeff,
                    const int it,
                    const int equivalent,
                    const double multiplier,
                    double* Value,
                    double* Value_equiv,
                    int* Default,
                    int* Renew,
                    int* Policya,
                    int* Policym,
                    int* Policyh,
                    int* Policyr,
                    int* Policyl,
                    double* Policyc,
                    double* Pricing_guess){

  int ind;
  int ind1;
  int ind2;
  int indsubs;
  
  double VV     = 0.0;
  double VV_eq  = 0.0;

  // Value normal
  double VVk    = powf(-10,11);  double VVn    = powf(-10,11);  double VVd    = powf(-10,11);
  double Vexk   = 0.0;           double Vexn   = 0.0;           double Vexd   = 0.0;

  // Value de consumption equivalent
  double VVk_eq    = powf(-10,11);  double VVn_eq    = powf(-10,11);  double VVd_eq    = powf(-10,11);
  double Vexk_eq   = 0.0;           double Vexn_eq   = 0.0;           double Vexd_eq   = 0.0;

  double cck    = 0;             double ccn    = 0;             double ccd    = 0;    // Consumption
  
  int hhk       = 0;             int hhn       = 0;             int hhd       = 0;    // Home ownership
  int hrk       = 0;             int hrn       = 0;             int hrd       = 0;    // Home renting
  int mmk       = 0;             int mmn       = 0;             int mmd       = 0;    // Mortgage
  int aak       = 0;             int aan       = 0;             int aad       = 0;    // Savings
  int llk       = 0;             int lln       = 0;             int lld       = 0;    // Labor
    
  double yy;
  double aa;
  double ll;
  double aaprime;
  double mm;
  double mmprime;
  double hh;
  double hhprime;
  double hhrent;
  double ddelta;
  double cons;
  double pprice;
  double mort_received;
  double mortgage_subsidy;
  double refinance_cost;

  // State variables that are parallelized
  // const int im  = blockIdx.x * blockDim.x + threadIdx.x;
  const int im  = blockIdx.x;
  const int ia  = blockIdx.y;
  const int id  = threadIdx.x;
  const int ih  = threadIdx.y;
  const int iy  = threadIdx.z;
  
  aa      = agrid[ia];
  mm      = mgrid[im];
  hh      = hgrid[ih];
  ddelta  = dgrid[id];

  ind     = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
  indsubs = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;
  
  mortgage_subsidy = mortsubsidy[indsubs];

  // Control variables
  for(int il=0; il<nl; il++){
    for(int iap=0; iap<na; iap++){
      for(int ihre=0; ihre<nr; ihre++){

        ll      = lgrid[il];
        aaprime = agrid[iap];
        hhrent  = rgrid[ihre];
        
        if(it < Tretirement){
          yy = ygrid[iy]*eprocess[it]*ll*(1-sstax-ltax);
        } else{
          yy = ygrid[iy]*pension;
        }

        // Keeping the same mortgage
        cons = aa + q*hh + yy*(1-incshock[it]) - mm - q*hhrent - Pa*aaprime - Ph*(ddelta + ddeltabar)*hh - lumpsum;
        
        Vexk       = 0.0;
        Vexk_eq    = 0.0;
        if(it < T-1){
          for(int idp=0; idp<nd; idp++){

            if(it < Tretirement){  // Income uncertainty before retirement
              for(int iyp=0; iyp<ny; iyp++){
                
                ind1 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + im*na + iap;
                ind2 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + 0*na + iap;
                
                Vexk = Vexk + P[iy*ny+iyp]*(1/(double)nd)*(rrho*Value[ind1] +  // Keeps mortg
                                                        (1-rrho)*Value[ind2]);    // Mortg disappears

                Vexk_eq = Vexk_eq + P[iy*ny+iyp]*(1/(double)nd)*(rrho*Value_equiv[ind1] +  // Keeps mortg
                                                               (1-rrho)*Value_equiv[ind2]);    // Mortg disappears
              }
            } else{   // Certainty after retirement
              
              ind1 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + im*na + iap;
              ind2 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + 0*na + iap;
              
              Vexk = Vexk + (1/(double)nd)*(rrho*Value[ind1] +  // Keeps mortg
                                            (1-rrho)*Value[ind2]);    // Mortg disappears

              Vexk_eq = Vexk_eq + (1/(double)nd)*(rrho*Value_equiv[ind1] +  // Keeps mortg
                                                  (1-rrho)*Value_equiv[ind2]);    // Mortg disappears
            }
          }
        }
        
        VV    = u(cons, hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexk;

        if(equivalent == 1){
          VV_eq = u(cons*(1+multiplier), hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexk_eq;
        }

        if(VV > VVk){
          VVk    = VV;
          VVk_eq = VV_eq;
          hhk    = ih;
          hrk    = ihre;
          mmk    = im;
          aak    = iap;
          cck    = cons;
          llk    = il;
        }
        
        // Defaulting => Household loses savings
        cons = maximumab(aa - rec_probab*((1+repay_coeff[it])*mm - Ph*(1-ddelta - ddeltabar)*hh*(1-sunk)), 0) + yy*(1-incshock[it]) - q*hhrent - Pa*aaprime - lumpsum;

        Vexd       = 0.0;
        Vexd_eq    = 0.0;
        if(it < T-1){
          for(int idp=0; idp<nd; idp++){

            if(it < Tretirement){  // Income uncertainty before retirement
              for(int iyp=0; iyp<ny; iyp++){
                ind1 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + 0*nm*na + 0*na + iap;

                Vexd = Vexd + P[iy*ny+iyp]*(1/(double)nd)*Value[ind1];

                Vexd_eq = Vexd_eq + P[iy*ny+iyp]*(1/(double)nd)*Value_equiv[ind1];
              }
            } else{   // Certainty after retirement
              ind1 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + 0*nm*na + 0*na + iap;
              
              Vexd = Vexd + (1/(double)nd)*Value[ind1];

              Vexd_eq = Vexd_eq + (1/(double)nd)*Value_equiv[ind1];
            }
          }
        }
        
        VV    = u(cons, hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexd;

        if(equivalent == 1){
          VV_eq = u(cons*(1+multiplier), hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexd_eq;
        }

        if(VV > VVd){
          VVd    = VV;
          VVd_eq = VV_eq;
          hhd    = 0;
          hrd    = ihre;
          mmd    = 0;
          aad    = iap;
          ccd    = cons;
          lld    = il;
        }
          
        // New mortgage
        for(int imp=0; imp<nm; imp++){
          for(int ihp=0; ihp<nh; ihp++){
            
            if(im == 0){
              if(imp > 0){
                refinance_cost = fcost;      // Issuing new mortgage
              } else{
                refinance_cost = 0.0;
              }
            } else{
              if(imp > 0){
                refinance_cost = refcost;  // Refinancing mortgage
              } else{
                refinance_cost = 0.0;      // Paying total debt
              }
            }

            mmprime      = mgrid[imp];
            hhprime      = hgrid[ihp];
            
            ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ihp*nm*na + imp*na + iap;
            pprice = Pricing_guess[ind];

            mort_received = mortg_function(mmprime, pprice, oomega, hhprime, Ph, repay_coeff[it]);
            
            cons = aa + Ph*(1-ddelta - ddeltabar)*hh + q*hhprime + yy*(1-incshock[it]) + mort_received - refinance_cost*(1+repay_coeff[it])*mmprime + mortgage_subsidy - (1+repay_coeff[it])*mm - q*hhrent - Ph*hhprime - Pa*aaprime - lumpsum;
            
            Vexn       = 0.0;
            Vexn_eq    = 0.0;
            if(it < T-1){
              for(int idp=0; idp<nd; idp++){

                if(it < Tretirement){  // Income uncertainty before retirement
                  for(int iyp=0; iyp<ny; iyp++){
                    
                    ind1 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + imp*na + iap;
                    ind2 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + 0*na + iap;
                    
                    Vexn = Vexn + P[iy*ny+iyp]*(1/(double)nd)*((rrho * Value[ind1]) + 
                                                            ((1-rrho) * Value[ind2]));

                    Vexn_eq = Vexn_eq + P[iy*ny+iyp]*(1/(double)nd)*((rrho * Value_equiv[ind1]) + 
                                                                     ((1-rrho) * Value_equiv[ind2]));
                  }
                } else{   // Certainty after retirement
                  ind1 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + imp*na + iap;
                  ind2 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + 0*na + iap;
                  
                  Vexn = Vexn + (1/(double)nd)*((rrho * Value[ind1]) + 
                                                ((1-rrho) * Value[ind2]));

                  Vexn_eq = Vexn_eq + (1/(double)nd)*((rrho * Value_equiv[ind1]) + 
                                                      ((1-rrho) * Value_equiv[ind2]));
                }
              }
            }
            
            VV    = u(cons, hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexn;

            if(equivalent == 1){
              VV_eq = u(cons*(1+multiplier), hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexn_eq;
            }

            if(VV > VVn){
              VVn    = VV;
              VVn_eq = VV_eq;
              hhn    = ihp;
              hrn    = ihre;
              mmn    = imp;
              aan    = iap;
              ccn    = cons;
              lln    = il;
            }
          }
        }
      }
    }
  }

  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
    
  if((VVk >= VVd) & (VVk >= VVn)){
    Value[ind]      = VVk;
    Value_equiv[ind]= VVk_eq;
    Policya[ind]    = aak;
    Policyh[ind]    = hhk;
    Policyr[ind]    = hrk;
    Policyl[ind]    = llk;
    Policym[ind]    = mmk;
    Policyc[ind]    = cck;
    Default[ind]    = 0;
    Renew[ind]      = 0;
  } else if((VVd > VVk) & (VVd > VVn)){
    Value[ind]      = VVd;
    Value_equiv[ind]= VVd_eq;
    Policya[ind]    = aad;
    Policyh[ind]    = hhd;
    Policyr[ind]    = hrd;
    Policyl[ind]    = lld;
    Policym[ind]    = mmd;
    Policyc[ind]    = ccd;
    Default[ind]    = 1;
    Renew[ind]      = 0;
  } else{
    Value[ind]      = VVn;
    Value_equiv[ind]= VVn_eq;
    Policya[ind]    = aan;
    Policyh[ind]    = hhn;
    Policyr[ind]    = hrn;
    Policyl[ind]    = lln;
    Policym[ind]    = mmn;
    Policyc[ind]    = ccn;
    Default[ind]    = 0;
    Renew[ind]      = 1;
  }

}



//==================================================================
//     Value Function Computation with different continuation
//==================================================================

__global__ void vfi_continuation(const int T, const int Tretirement, const int na, const int nm, const int nh, const int nr, const int nl,
                                  const int nd,    const int ny, const int uti,
                                  const double rrho,    const double bbeta, const double Ph_today, const double q,  const double Pa, const double ddeltabar_today,
                                  const double ssigma,  const double ppsi,  const double kkappa, const double tthetalab, const double eetalab,
                                  const double fcost, const double refcost, const double pension, const double sstax, const double ltax,
                                  const double lumpsum, const double oomega, const double rec_probab, const double sunk,
                                  const double *incshock, const double *mortsubsidy,
                                  const double *agrid, 
                                  const double *mgrid, 
                                  const double *hgrid, 
                                  const double *rgrid, 
                                  const double *lgrid, 
                                  const double *dgrid, 
                                  const double *ygrid, 
                                  const double *P, 
                                  const double *eprocess, 
                                  const double *survival,
                                  const double *repay_coeff,
                                  const int it,
                                  const int equivalent,
                                  const double multiplier,
                                  const double* Value_future,
                                  const double* Value_equiv_future,
                                  double* Value,
                                  double* Value_equiv,
                                  int* Default,
                                  int* Renew,
                                  int* Policya,
                                  int* Policym,
                                  int* Policyh,
                                  int* Policyr,
                                  int* Policyl,
                                  double* Policyc,
                                  double* Pricing_guess){
  
  int ind;
  int ind1;
  int ind2;
  int indsubs;
  
  double VV     = 0.0;
  double VV_eq  = 0.0;

  double VVk    = powf(-10,11);  double VVn    = powf(-10,11);  double VVd    = powf(-10,11);
  double Vexk   = 0.0;           double Vexn   = 0.0;           double Vexd   = 0.0;

  double VVk_eq    = powf(-10,11);  double VVn_eq    = powf(-10,11);  double VVd_eq    = powf(-10,11);
  double Vexk_eq   = 0.0;           double Vexn_eq   = 0.0;           double Vexd_eq   = 0.0;

  double cck    = 0;             double ccn    = 0;             double ccd    = 0;    // Consumption
  
  int hhk    = 0;  int hhn    = 0;  int hhd    = 0;    // Home ownership
  int hrk    = 0;  int hrn    = 0;  int hrd    = 0;    // Home renting
  int mmk    = 0;  int mmn    = 0;  int mmd    = 0;    // Mortgage
  int aak    = 0;  int aan    = 0;  int aad    = 0;    // Savings
  int llk    = 0;  int lln    = 0;  int lld    = 0;    // Labor

  double yy;
  double aa;
  double ll;
  double aaprime;
  double mm;
  double mmprime;
  double hh;
  double hhprime;
  double hhrent;
  double ddelta;
  double cons;
  double pprice;
  double mort_received;
  double mortgage_subsidy;
  double refinance_cost;

  // State variables that are parallelized
  // const int im  = blockIdx.x * blockDim.x + threadIdx.x;
  const int im  = blockIdx.x;
  const int ia  = blockIdx.y;
  const int id  = threadIdx.x;
  const int ih  = threadIdx.y;
  const int iy  = threadIdx.z;

  aa      = agrid[ia];
  mm      = mgrid[im];
  hh      = hgrid[ih];
  ddelta  = dgrid[id];
 
  ind     = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
  indsubs = it*ny*nh*nm*na + iy*nh*nm*na + ih*nm*na + im*na + ia;
  
  mortgage_subsidy = mortsubsidy[indsubs];


  // Control variables
  for(int il=0; il<nl; il++){
    for(int iap=0; iap<na; iap++){
      for(int ihre=0; ihre<nr; ihre++){

        ll      = lgrid[il];
        aaprime = agrid[iap];
        hhrent  = rgrid[ihre];

        if(it < Tretirement){
          yy = ygrid[iy]*eprocess[it]*ll*(1-sstax-ltax);
        } else{
          yy = ygrid[iy]*pension;
        }

        // Keeping the same mortgage
        cons = aa + q*hh + yy*(1-incshock[it]) - mm - q*hhrent - Pa*aaprime - Ph_today*(ddelta + ddeltabar_today)*hh - lumpsum;
        
        Vexk       = 0.0;
        Vexk_eq       = 0.0;
        if(it < T-1){
          for(int idp=0; idp<nd; idp++){

            if(it < Tretirement){  // Income uncertainty before retirement
              for(int iyp=0; iyp<ny; iyp++){
                
                ind1 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + im*na + iap;
                ind2 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + 0*na + iap;
                
                Vexk = Vexk + P[iy*ny+iyp]*(1/(double)nd)*(rrho*Value_future[ind1] +  // Keeps mortg
                                                        (1-rrho)*Value_future[ind2]);    // Mortg disappears

                Vexk_eq = Vexk_eq + P[iy*ny+iyp]*(1/(double)nd)*(rrho*Value_equiv_future[ind1] +  // Keeps mortg
                                                                (1-rrho)*Value_equiv_future[ind2]);    // Mortg disappears
              }
            } else{   // Certainty after retirement
              
              ind1 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + im*na + iap;
              ind2 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ih*nm*na + 0*na + iap;
              
              Vexk = Vexk + (1/(double)nd)*(rrho*Value_future[ind1] +  // Keeps mortg
                                            (1-rrho)*Value_future[ind2]);    // Mortg disappears

              Vexk_eq = Vexk_eq + (1/(double)nd)*(rrho*Value_equiv_future[ind1] +  // Keeps mortg
                                                 (1-rrho)*Value_equiv_future[ind2]);    // Mortg disappears
            }
          }
        }
        
        VV    = u(cons, hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexk;
        
        if(equivalent == 1){
          VV_eq = u(cons*(1+multiplier), hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexk_eq;
        }
        
        if(VV > VVk){
          VVk    = VV;
          VVk_eq = VV_eq;
          hhk    = ih;
          hrk    = ihre;
          mmk    = im;
          aak    = iap;
          cck    = cons;
          llk    = il;
        }
        
        // Defaulting => Household loses savings
        cons = maximumab(aa - rec_probab*((1+repay_coeff[it])*mm - Ph_today*(1-ddelta - ddeltabar_today)*hh*(1-sunk)), 0) + yy*(1-incshock[it]) - q*hhrent - Pa*aaprime - lumpsum;
        
        Vexd       = 0.0;
        Vexd_eq       = 0.0;
        if(it < T-1){
          for(int idp=0; idp<nd; idp++){

            if(it < Tretirement){  // Income uncertainty before retirement
              for(int iyp=0; iyp<ny; iyp++){
                ind1 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + iap;
  
                Vexd = Vexd + P[iy*ny+iyp]*(1/(double)nd)*Value_future[ind1];
                Vexd_eq = Vexd_eq + P[iy*ny+iyp]*(1/(double)nd)*Value_equiv_future[ind1];
              }
            } else{   // Certainty after retirement
              ind1 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + iap;
              
              Vexd = Vexd + (1/(double)nd)*Value_future[ind1];
              Vexd_eq = Vexd_eq + (1/(double)nd)*Value_equiv_future[ind1];
            }
          }
        }
        
        VV    = u(cons, hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexd;

        if(equivalent == 1){
          VV_eq = u(cons*(1+multiplier), hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexd_eq;
        }

        if(VV > VVd){
          VVd    = VV;
          VVd_eq = VV_eq;
          hhd    = 0;
          hrd    = ihre;
          mmd    = 0;
          aad    = iap;
          ccd    = cons;
          lld    = il;
        }
          
        // New mortgage
        for(int imp=0; imp<nm; imp++){
          for(int ihp=0; ihp<nh; ihp++){
            
            if(im == 0){
              if(imp > 0){
                refinance_cost = fcost;      // Issuing new mortgage
              } else{
                refinance_cost = 0.0;
              }
            } else{
              if(imp > 0){
                refinance_cost = refcost;  // Refinancing mortgage
              } else{
                refinance_cost = 0.0;      // Paying total debt
              }
            }
  
            mmprime      = mgrid[imp];
            hhprime      = hgrid[ihp];
            
            ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ihp*nm*na + imp*na + iap;
            pprice = Pricing_guess[ind];
            
            mort_received = mortg_function(mmprime, pprice, oomega, hhprime, Ph_today, repay_coeff[it]);
  
            cons = aa + Ph_today*(1-ddelta - ddeltabar_today)*hh + q*hhprime + yy*(1-incshock[it]) - refinance_cost*(1+repay_coeff[it])*mmprime + mort_received + mortgage_subsidy - (1+repay_coeff[it])*mm - q*hhrent - Ph_today*hhprime - Pa*aaprime - lumpsum;
            
            Vexn       = 0.0;
            Vexn_eq    = 0.0;
            if(it < T-1){
              for(int idp=0; idp<nd; idp++){

                if(it < Tretirement){  // Income uncertainty before retirement
                  for(int iyp=0; iyp<ny; iyp++){
                    
                    ind1 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + imp*na + iap;
                    ind2 = (it+1)*ny*nd*nh*nm*na + iyp*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + 0*na + iap;
                    
                    Vexn = Vexn + P[iy*ny+iyp]*(1/(double)nd)*((rrho * Value_future[ind1]) + 
                                                            ((1-rrho) * Value_future[ind2]));

                    Vexn_eq = Vexn_eq + P[iy*ny+iyp]*(1/(double)nd)*((rrho * Value_equiv_future[ind1]) + 
                                                                    ((1-rrho) * Value_equiv_future[ind2]));
                  }
                } else{   // Certainty after retirement
                  ind1 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + imp*na + iap;
                  ind2 = (it+1)*ny*nd*nh*nm*na + iy*nd*nh*nm*na + idp*nh*nm*na + ihp*nm*na + 0*na + iap;
                  
                  Vexn = Vexn + (1/(double)nd)*((rrho * Value_future[ind1]) + 
                                                ((1-rrho) * Value_future[ind2]));

                  Vexn_eq = Vexn_eq + (1/(double)nd)*((rrho * Value_equiv_future[ind1]) + 
                                                      ((1-rrho) * Value_equiv_future[ind2]));
                }
              }
            }
            
            VV    = u(cons, hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexn;

            if(equivalent == 1){
              VV_eq = u(cons*(1+multiplier), hhrent, ll, ssigma, ppsi, uti, kkappa, tthetalab, eetalab) + bbeta*survival[it]*Vexn_eq;
            }
            
            if(VV > VVn){
              VVn    = VV;
              VVn_eq = VV_eq;
              hhn    = ihp;
              hrn    = ihre;
              mmn    = imp;
              aan    = iap;
              ccn    = cons;
              lln    = il;
            }
          }
        }
      }
    }
  }

  ind = it*ny*nd*nh*nm*na + iy*nd*nh*nm*na + id*nh*nm*na + ih*nm*na + im*na + ia;
    
  if((VVk >= VVd) & (VVk >= VVn)){
    Value[ind]      = VVk;
    Value_equiv[ind]= VVk_eq;
    Policya[ind]    = aak;
    Policyh[ind]    = hhk;
    Policyr[ind]    = hrk;
    Policyl[ind]    = llk;
    Policym[ind]    = mmk;
    Policyc[ind]    = cck;
    Default[ind]    = 0;
    Renew[ind]      = 0;
  } else if((VVd > VVk) & (VVd >= VVn)){
    Value[ind]      = VVd;
    Value_equiv[ind]= VVd_eq;
    Policya[ind]    = aad;
    Policyh[ind]    = hhd;
    Policyr[ind]    = hrd;
    Policyl[ind]    = lld;
    Policym[ind]    = mmd;
    Policyc[ind]    = ccd;
    Default[ind]    = 1;
    Renew[ind]      = 0;
  } else{
    Value[ind]      = VVn;
    Value_equiv[ind]= VVn_eq;
    Policya[ind]    = aan;
    Policyh[ind]    = hhn;
    Policyr[ind]    = hrn;
    Policyl[ind]    = lln;
    Policym[ind]    = mmn;
    Policyc[ind]    = ccn;
    Default[ind]    = 0;
    Renew[ind]      = 1;
  }

}